#include "utils.h"

void allocateToDevice(float **x, int size)
{
    hipError_t result = hipMalloc((void **)x, size);
    CUDA_CHECK(result);
}

void freeFromDevice(float *x)
{
    hipError_t result = hipFree(x);
    CUDA_CHECK(result);
}

void copyFromHostToDevice(float *dest, float *src, int size)
{
    hipError_t result = hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
    CUDA_CHECK(result);
}

void copyFromDeviceToHost(float *dest, float *src, int size)
{
    hipError_t result = hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
    CUDA_CHECK(result);
}
