#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>


void allocateToDevice(float* x, int size) {
    hipError_t error = hipMalloc((void**) &x, size);
    if(error != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(error), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}

void freeFromDevice(float* x) {
    hipError_t error = hipFree(x);
    if(error != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(error), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}

void copyFromHostToDevice(float* dest, float* src, int size) {
    hipError_t error = hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
    if(error != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(error), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
}


void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
    int size = n * sizeof(float);
    float *A_d, *B_d, *C_d;

    // allocate memory on the device
    allocateToDevice(A_d, size);
    allocateToDevice(B_d, size);
    allocateToDevice(C_d, size);

    // copy from host to device

    // dest, src, size, direction
    copyFromHostToDevice(A_d, A_h, size);
    copyFromHostToDevice(B_d, B_h, size);
    copyFromHostToDevice(C_d, C_h, size);


    // free memory
    freeFromDevice(A_d);
    freeFromDevice(B_d);
    freeFromDevice(C_d);
}

int main() {

    
    return 0;
}
